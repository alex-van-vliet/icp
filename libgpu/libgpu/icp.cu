#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>

#include "cuda/memory.hh"
#include "icp.hh"
#include "matrix.hh"
#include "svd.hh"
#include "vp-tree.hh"

namespace libgpu
{
    __global__ void to_transformation_kernel(GPUMatrix rotation,
                                             GPUMatrix translation,
                                             GPUMatrix res)
    {
        uint i = threadIdx.x;
        uint j = threadIdx.y;

        if (i < 3)
        {
            if (j < 3)
                res(i, j) = rotation(i, j);
            else // i < 3 && j == 3
                res(i, 3) = translation(0, i);
        }
        else // i == 3
        {
            if (j < 3)
                res(3, j) = 0;
            else // i == 3 && j == 3
                res(3, 3) = 1;
        }
    }

    GPUMatrix to_transformation(const GPUMatrix& rotation,
                                const GPUMatrix& translation)
    {
        assert(rotation.rows == 3);
        assert(rotation.cols == 3);
        assert(translation.rows == 1);
        assert(translation.cols == 3);

        GPUMatrix transformation(4, 4);

        dim3 blockdim(4, 4);
        to_transformation_kernel<<<1, blockdim>>>(rotation, translation,
                                                  transformation);

        return transformation;
    }

    GPUMatrix find_alignment(const GPUMatrix& p_centered, const GPUMatrix& mu_p,
                             const GPUMatrix& y, const GPUMatrix& mu_m)
    {
        auto covariance = GPUMatrix::find_covariance(p_centered, y);

        auto rotation = GPUMatrix::from_cpu(find_rotation(covariance.to_cpu()));

        auto translation = mu_m.subtract(mu_p.dot(rotation.transpose()));

        return to_transformation(rotation, translation);
    }

    __global__ void compute_error_kernel(GPUMatrix m, GPUMatrix p,
                                         GPUMatrix mu_m, GPUMatrix diffs)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= m.rows)
            return;

        assert(m.cols == p.cols);
        assert(m.cols == mu_m.cols);
        assert(mu_m.rows == 1);

        float dist = 0;
        for (size_t j = 0; j < m.cols; ++j)
        {
            float diff = m(i, j) + mu_m(0, j) - p(i, j);
            dist += diff * diff;
        }

        diffs(i, 0) = dist;
    }

    __global__ void compute_error_reduce_kernel(GPUMatrix diffs, float* error_d)
    {
        float error = 0;

        for (size_t i = 0; i < diffs.rows; ++i)
            error += diffs(i, 0);

        *error_d = error;
    }

    float compute_error(const GPUMatrix& m, const GPUMatrix& p,
                        const GPUMatrix& mu_m)
    {
        GPUMatrix diffs(m.rows, 1);

        dim3 blockdim(1024);
        dim3 griddim((m.rows + blockdim.x - 1) / blockdim.x);
        compute_error_kernel<<<griddim, blockdim>>>(m, p, mu_m, diffs);

        auto error_d = cuda::malloc<float>(1);

        compute_error_reduce_kernel<<<1, 1>>>(diffs, error_d.get());

        float error = 0;
        hipMemcpy(&error, error_d.get(), sizeof(float),
                   hipMemcpyDeviceToHost);

        return error;
    }

    __global__ void apply_alignment_kernel(GPUMatrix p,
                                           GPUMatrix transformation)
    {
        uint i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i >= p.rows)
            return;

        float values[3] = {0};
        for (size_t j = 0; j < p.cols; ++j)
        {
            for (size_t k = 0; k < 3; ++k)
                values[j] += p(i, k) * transformation(j, k);
            values[j] += transformation(j, 3);
        }

        for (size_t j = 0; j < p.cols; ++j)
            p(i, j) = values[j];
    }

    void apply_alignment(GPUMatrix& p, const GPUMatrix& transformation)
    {
        assert(p.cols == 3);
        assert(transformation.rows == 4);
        assert(transformation.cols == 4);

        dim3 blockdim(1024);
        dim3 griddim((p.rows + blockdim.x - 1) / blockdim.x);
        apply_alignment_kernel<<<griddim, blockdim>>>(p, transformation);
    }

    std::tuple<utils::Matrix<float>, libcpu::point_list>
    icp(const libcpu::point_list& m_cpu, const libcpu::point_list& p,
        size_t iterations, float threshold, uint vp_threshold)
    {
        auto new_p = GPUMatrix::from_point_list(p);
        auto m = GPUMatrix::from_point_list(m_cpu);

        auto transformation = GPUMatrix::eye(4);

        float error = std::numeric_limits<float>::infinity();

        auto mu_m = m.mean();

        auto tree = GPUVPTree::from_cpu(libcpu::VPTree(
            vp_threshold, m.subtract_rowwise(mu_m).to_point_list()));

        for (size_t i = 0; i < iterations && error > threshold; ++i)
        {
            std::cerr << "Starting iter " << (i + 1) << "/" << iterations
                      << std::endl;
            auto mu_p = new_p.mean();
            auto p_centered = new_p.subtract_rowwise(mu_p);
            auto y = tree.closest(p_centered);

            auto new_transformation = find_alignment(p_centered, mu_p, y, mu_m);

            transformation = new_transformation.dot(transformation);
            apply_alignment(new_p, new_transformation);
            error = compute_error(y, new_p, mu_m);
            std::cerr << "Error: " << error << std::endl;
        }

        return {transformation.to_cpu(), new_p.to_point_list()};
    }
} // namespace libgpu